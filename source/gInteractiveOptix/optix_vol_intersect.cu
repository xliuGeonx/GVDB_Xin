#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017, NVIDIA Corporation
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//----------------------------------------------------------------------------------

#include "optix_extra_math.cuh"			
#include ""

//------------------- GVDB Structure
#define OPTIX_PATHWAY
#include "cuda_gvdb_scene.cuh"		// GVDB Scene
#include "cuda_gvdb_nodes.cuh"		// GVDB Node structure
#include "cuda_gvdb_geom.cuh"		// GVDB Geom helpers
#include "cuda_gvdb_dda.cuh"		// GVDB DDA 
#include "cuda_gvdb_raycast.cuh"	// GVDB Raycasting
//--------------------


rtBuffer<float3>		  brick_buffer;

rtDeclareVariable(uint,	  mat_id, , );
rtDeclareVariable(float3, light_pos, , );

rtDeclareVariable(float3, back_hit_point,	attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,	attribute front_hit_point, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,	attribute shading_normal, ); 
rtDeclareVariable(float4, deep_color,		attribute deep_color, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

struct PerRayData_radiance
{
	float3	result;
	float	length; 
	float	alpha;
	int		depth;
	int		rtype;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

//------ Intersection Program

RT_PROGRAM void vol_intersect( int primIdx )
{
	float3 hit = make_float3(NOHIT,NOHIT,NOHIT);	
	float3 norm = make_float3(0,0,0);
	float4 clr = make_float4(0,0,0,0);	
	float t;

	//-- Ray march		
	float4 hclr;
	rayCast ( SCN_SHADE, gvdb.top_lev, 0, ray.origin, ray.direction, hit, norm, hclr, raySurfaceBrick );
	if ( hit.z == NOHIT) return;	
	t = length ( hit - ray.origin );

	// report intersection to optix
	if ( rtPotentialIntersection( t ) ) {	

		shading_normal = norm;		
		geometric_normal = norm;
		front_hit_point = hit + shading_normal*gvdb.voxelsize;
		back_hit_point  = hit - shading_normal*gvdb.voxelsize*5;
		deep_color = make_float4(1,1,1,1);
		if ( prd_radiance.rtype == SHADOW_RAY ) deep_color.w = (hit.x==NOHIT) ? 1 : 0;

		rtReportIntersection( mat_id );
	}
}

RT_PROGRAM void vol_deep( int primIdx )
{
	float3 hit = make_float3(NOHIT,NOHIT,NOHIT);	
	float3 norm = make_float3(0,1,0);
	float4 clr = make_float4(0,0,0,1);	
	if ( prd_radiance.rtype == MESH_RAY ) return;	

	// ---- Debugging
	// Uncomment this code to demonstrate tracing of the bounding box 
	// surrounding the volume.
	/*hit = rayBoxIntersect ( ray.origin, ray.direction, gvdb.bmin, gvdb.bmax );
	if ( hit.z == NOHIT ) return;
	if ( rtPotentialIntersection ( hit.x ) ) {
		shading_normal = norm;		
		geometric_normal = norm;
		front_hit_point = ray.origin + hit.x * ray.direction;
		back_hit_point  = ray.origin + hit.y * ray.direction;
		deep_color = make_float4( front_hit_point/200.0, 0.5);	
		rtReportIntersection( 0 );		
	}
	return;*/

	//-- Raycast
	rayCast ( SHADE_VOLUME, gvdb.top_lev, 0, ray.origin, ray.direction, hit, norm, clr, rayDeepBrick );
	if ( hit.z == NOHIT) return;	

	if ( rtPotentialIntersection( hit.x ) ) {

		shading_normal = norm;		
		geometric_normal = norm;
		front_hit_point = ray.origin + hit.x * ray.direction;
		back_hit_point  = ray.origin + hit.y * ray.direction;
		deep_color = make_float4 ( fxyz(clr), 1.0-clr.w );

		rtReportIntersection( 0 );			
	}
}

RT_PROGRAM void vol_levelset ( int primIdx )
{
	float3 hit = make_float3(NOHIT,1,1);	
	float3 norm = make_float3(0,0,0);
	float4 clr = make_float4(0,0,0,0);	
	float t;

	//-- Ray march		
	rayCast ( 0, gvdb.top_lev, 0, ray.origin, ray.direction, hit, norm, clr, rayLevelSetBrick );
	if ( hit.x == NOHIT) return;	
	t = length ( hit - ray.origin );

	// report intersection to optix
	if ( rtPotentialIntersection( t ) ) {	

		shading_normal = norm;		
		geometric_normal = norm;
		front_hit_point = hit + shading_normal*gvdb.voxelsize;
		back_hit_point  = hit - shading_normal*gvdb.voxelsize*5;
		deep_color = make_float4(1,1,1,1);
		if ( prd_radiance.rtype == SHADOW_RAY ) deep_color.w = (hit.x==NOHIT) ? 1 : 0;

		rtReportIntersection( mat_id );
	}
}


RT_PROGRAM void vol_bounds (int primIdx, float result[6])
{
	// AABB bounds is just the brick extents	
	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = brick_buffer[ primIdx*2 ];
	aabb->m_max = brick_buffer[ primIdx*2+1 ];
}

